#include "hip/hip_runtime.h"
#include "stdafx.h"

__constant__ int iTestIndices[iMaxNumberOfTrainedElements];

// Old executing kernel
__global__ void executeLayerKernel_OLD(const real_gpu *dp_pLayerInput,const real_gpu *dp_pWeights,real_gpu *dp_pLayerOutput,real_gpu *dp_pDerivativeOfLastOutput,int p_iNumInputNeurons, Neuron::NeuronType p_eNeuronType)
{
	int iNumInputNeuronsAligned = ALIGN_UP(p_iNumInputNeurons, HALF_WARP);
	int iNumOutputNeuronsAligned = ALIGN_UP(blockDim.x, HALF_WARP);

	const real_gpu *d_LayerInputThisTest = dp_pLayerInput + blockIdx.x*iNumInputNeuronsAligned;
	const real_gpu *d_WeightsThisTest = dp_pWeights + threadIdx.x*p_iNumInputNeurons;
	real_gpu *d_pLayerOutputThisTest = dp_pLayerOutput + blockIdx.x*iNumOutputNeuronsAligned + threadIdx.x;
	real_gpu *d_pDerivativeOfLastOutputThisTest = dp_pDerivativeOfLastOutput + blockIdx.x*iNumOutputNeuronsAligned + threadIdx.x;

	real_gpu dResult = 0.0f;

	for(int iWeightIndex = 0;iWeightIndex < p_iNumInputNeurons; ++iWeightIndex)
	{
		PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d , iWeightIndex %d : d_LayerInputThisTest %f , d_WeightsThisTest %f , MULT %f\n",blockIdx.x,threadIdx.x,iWeightIndex,d_LayerInputThisTest[iWeightIndex],d_WeightsThisTest[iWeightIndex],d_LayerInputThisTest[iWeightIndex] * d_WeightsThisTest[iWeightIndex]);
		dResult += d_LayerInputThisTest[iWeightIndex] * d_WeightsThisTest[iWeightIndex];
	}

	real_gpu dDerivativeOfLastOutput = 0.0f;

	PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d : dResult before output function %f\n",blockIdx.x,threadIdx.x,dResult);

	switch(p_eNeuronType)
	{
		case Neuron::NT_LINEAR:
		{
			dDerivativeOfLastOutput = 1.0f;
			break;	// Do nothing
		}
		case Neuron::NT_SIGMOID:
		{
			real_gpu dExp = exp(-dResult);
			dResult = 1.0f / (1.0f + dExp);
			dDerivativeOfLastOutput = dExp / pow(1.0f + dExp,2);
			break;
		}
		case Neuron::NT_TANH:
		{
			dResult = tanhf(dResult);
			dDerivativeOfLastOutput = 1 - tanhf(dResult);
			break;
		}
	}

	if(threadIdx.x == blockDim.x - 1)
		dResult = 1.0f; /* bias */

	*d_pLayerOutputThisTest = dResult;

	// We only need derivative of last output if we are in training!
	if(dp_pDerivativeOfLastOutput != NULL)
		*d_pDerivativeOfLastOutputThisTest = dDerivativeOfLastOutput;

	PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d : first d_LayerInputThisTest %f , first d_WeightsThisTest %f , dResult %f , dDerivativeOfLastOutput %f\n",blockIdx.x,threadIdx.x,d_LayerInputThisTest[0],d_WeightsThisTest[0],dResult,dDerivativeOfLastOutput);
}

extern "C" void executeLayerCUDA_OLD(const real_gpu *dp_pLayerInput,const real_gpu *dp_pWeights,real_gpu *dp_pLayerOutput,real_gpu *dp_pDerivativeOfLastOutput,int p_iTestCount,int p_iOutputNeuronCount,int p_iNumInputNeurons,Neuron::NeuronType p_eNeuronType)
{
	executeLayerKernel_OLD <<<p_iTestCount,p_iOutputNeuronCount+1>>> (dp_pLayerInput,dp_pWeights,dp_pLayerOutput,dp_pDerivativeOfLastOutput,p_iNumInputNeurons,p_eNeuronType);
}

// New executing kernel
__global__ void executeLayerKernel(const real_gpu *dp_pLayerInput,const real_gpu *dp_pWeights,real_gpu *dp_pLayerOutput,real_gpu *dp_pDerivativeOfLastOutput,int p_iNumInputNeurons
								   ,int p_iNumInputNeuronsAligned, Neuron::NeuronType p_eNeuronType,int p_iOutputNeuronCount,bool p_bInTraining,int p_iHowMuchMemoryForWeights,int p_iTestCount)
{
	extern __shared__ real_gpu s_InputNeurons[];
	real_gpu* s_InputNeurons2 = &s_InputNeurons[p_iNumInputNeurons];
	real_gpu* s_InputWeights = &s_InputNeurons2[p_iNumInputNeurons];

	int iTestIndex,iTestIndex2;
	if(p_bInTraining)
	{
		iTestIndex = iTestIndices[2*blockIdx.x];
		iTestIndex2 = iTestIndices[2*blockIdx.x+1];
	}
	else
	{
		iTestIndex = 2*blockIdx.x;
		iTestIndex2 = 2*blockIdx.x+1;
	}

	const real_gpu *d_LayerInputThisTest = dp_pLayerInput + iTestIndex*p_iNumInputNeuronsAligned;
	const real_gpu *d_LayerInputThisTest2 = dp_pLayerInput + iTestIndex2*p_iNumInputNeuronsAligned;
	int iMoveWeightsForThisTest = threadIdx.x*p_iNumInputNeurons;
	real_gpu *d_pLayerOutputThisTest = dp_pLayerOutput + (2*blockIdx.x)*blockDim.x + threadIdx.x;
	real_gpu *d_pLayerOutputThisTest2 = dp_pLayerOutput + (2*blockIdx.x+1)*blockDim.x + threadIdx.x;
	real_gpu *d_pDerivativeOfLastOutputThisTest = dp_pDerivativeOfLastOutput + (2*blockIdx.x)*blockDim.x + threadIdx.x;
	real_gpu *d_pDerivativeOfLastOutputThisTest2 = dp_pDerivativeOfLastOutput + (2*blockIdx.x+1)*blockDim.x + threadIdx.x;

#ifdef PRINT_DEBUG
	const real_gpu *d_WeightsThisTest = dp_pWeights + iMoveWeightsForThisTest;
#endif

	// first, we copy d_LayerInputThisTest to s_InputNeurons
	for(int iInputIndex = threadIdx.x;iInputIndex < p_iNumInputNeurons; iInputIndex+=blockDim.x)
	{
		s_InputNeurons[iInputIndex] = d_LayerInputThisTest[iInputIndex];
		s_InputNeurons2[iInputIndex] = d_LayerInputThisTest2[iInputIndex];
		PRINT_MEMORY_INFO(dp_pLayerInput,&d_LayerInputThisTest[iInputIndex]);
	}

	// we have to make sure that all data was written to shared memory
	__syncthreads();

	real_gpu dResult = 0.0f;
	real_gpu dResult2 = 0.0f;

	//if(threadIdx.x == 1 && blockIdx.x == 1)
	//{
	//	PRINT_DEBUG_INFO("BX %d TX %d | INPUT %d | WEIGHTS %d | OUTPUT %d\n",blockIdx.x,threadIdx.x,d_LayerInputThisTest - dp_pLayerInput,d_WeightsThisTest - dp_pWeights,d_pLayerOutputThisTest - dp_pLayerOutput);
	//}

	int iNumOfWeights = p_iNumInputNeurons * p_iOutputNeuronCount;
	int iNumOfWeightsAligned = ALIGN_UP(iNumOfWeights,blockDim.x);
	for(int iWeightIndex = threadIdx.x, iWeightIndexBase = 0 ; iWeightIndex < iNumOfWeightsAligned ; iWeightIndex += p_iHowMuchMemoryForWeights, iWeightIndexBase += p_iHowMuchMemoryForWeights)
	{
		/*if(threadIdx.x == 0)
		{
			PRINT_DEBUG_INFO("GPU: NEW BATCH!!!!!!!!! iWeightIndexBase = %d , blockDim.x = %d\n",iWeightIndexBase,blockDim.x);
		}*/

		// first, we copy d_WeightsThisTest to s_InputWeights (it is only a part of weights).
		// We don't have to use 'if(iWeightIndex < iNumOfWeights)', because memory for weights was already padded (now it's about 5% faster)
		for(int iCopiedWeightIndex = 0;iCopiedWeightIndex<p_iHowMuchMemoryForWeights;iCopiedWeightIndex += blockDim.x)
		{
			s_InputWeights[iCopiedWeightIndex + threadIdx.x] = dp_pWeights[iCopiedWeightIndex + iWeightIndex];
			PRINT_MEMORY_INFO(dp_pWeights,&dp_pWeights[iCopiedWeightIndex + iWeightIndex]);
		}

		__syncthreads(); // We make sure that all data was written to shared memory

		int iFirstElementInThisBatch = iMoveWeightsForThisTest - iWeightIndexBase;
		int iLastElementInThisBatch = iFirstElementInThisBatch + p_iNumInputNeurons;

		// Not all threads are used in calulations
		//PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d : iFirstElementInThisBatch %d , iLastElementInThisBatch %d , T1  = [%d] , T2 = [%d] , T3 = [%d]\n",blockIdx.x,threadIdx.x,iFirstElementInThisBatch,iLastElementInThisBatch,(threadIdx.x < p_iOutputNeuronCount),(iLastElementInThisBatch >= 0),(iFirstElementInThisBatch < 0 || iFirstElementInThisBatch < blockDim.x));
		if(threadIdx.x < p_iOutputNeuronCount && iLastElementInThisBatch >= 0 && (iFirstElementInThisBatch < 0 || iFirstElementInThisBatch < p_iHowMuchMemoryForWeights))
		{
			int iFirstWeightIndex = max(0,-iFirstElementInThisBatch);
			int iLastWeightIndex = min(p_iNumInputNeurons,p_iNumInputNeurons - (iLastElementInThisBatch - p_iHowMuchMemoryForWeights));
			for(int iWeightIndexToAdd = iFirstWeightIndex;iWeightIndexToAdd < iLastWeightIndex; ++iWeightIndexToAdd)
			{
				int iWeightIndexHere = iWeightIndexToAdd - iWeightIndexBase + iMoveWeightsForThisTest;
				//PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d , iWeightIndexToAdd %d : d_LayerInputThisTest %f , d_WeightsThisTest %f , iWeightIndexHere %d, val[%d] %f , MULT %f\n",blockIdx.x,threadIdx.x,iWeightIndexToAdd,d_LayerInputThisTest[iWeightIndexToAdd],d_WeightsThisTest[iWeightIndexToAdd],iWeightIndexHere,iWeightIndexHere,s_InputWeights[iWeightIndexHere],d_LayerInputThisTest[iWeightIndexToAdd] * d_WeightsThisTest[iWeightIndexToAdd]);

				dResult += s_InputNeurons[iWeightIndexToAdd] * s_InputWeights[iWeightIndexHere];
				dResult2 += s_InputNeurons2[iWeightIndexToAdd] * s_InputWeights[iWeightIndexHere];
			}
		}

		__syncthreads(); // We make sure that all data was read by all threads
	}

	if(threadIdx.x <= p_iOutputNeuronCount)
	{
		real_gpu dDerivativeOfLastOutput = 0.0f;
		real_gpu dDerivativeOfLastOutput2 = 0.0f;

		//PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d : dResult before output function %f\n",blockIdx.x,threadIdx.x,dResult);

		switch(p_eNeuronType)
		{
			case Neuron::NT_LINEAR:
			{
				dDerivativeOfLastOutput = 1.0f;
				dDerivativeOfLastOutput2 = 1.0f;
				break;	// Do nothing
			}
			case Neuron::NT_SIGMOID:
			{
				real_gpu dExp = __expf(-dResult);
				dResult = 1.0f / (1.0f + dExp);
				dDerivativeOfLastOutput = dExp / __powf(1.0f + dExp,2);
				real_gpu dExp2 = __expf(-dResult2);
				dResult2 = 1.0f / (1.0f + dExp2);
				dDerivativeOfLastOutput2 = dExp2 / __powf(1.0f + dExp2,2);
				break;
			}
			case Neuron::NT_TANH:
			{
				dResult = tanhf(dResult);
				dDerivativeOfLastOutput = 1 - tanhf(dResult);
				dResult2 = tanhf(dResult2);
				dDerivativeOfLastOutput2 = 1 - tanhf(dResult2);
				break;
			}
		}

		if(threadIdx.x == p_iOutputNeuronCount)
		{
			dResult = 1.0f; // bias
			dResult2 = 1.0f; // bias
		}

		// If there is an odd number of blocks, the values for last block will not be written
		bool bWriteSecondValue = (2*blockIdx.x != p_iTestCount-1);

		//PRINT_DEBUG_INFO("XXXXXXXXXXXXXXXXXXXXXXXXXXXXGPU: Test %d , Neuron %d : %d\n",blockIdx.x,threadIdx.x,blockIdx.x*iNumOutputNeuronsAligned + threadIdx.x);
		*d_pLayerOutputThisTest = dResult;

		if(bWriteSecondValue)
			*d_pLayerOutputThisTest2 = dResult2;

		PRINT_MEMORY_INFO(dp_pLayerOutput,d_pLayerOutputThisTest);

		// We only need derivative of last output if we are in training!
		if(dp_pDerivativeOfLastOutput != NULL)
		{
			*d_pDerivativeOfLastOutputThisTest = dDerivativeOfLastOutput;

			if(bWriteSecondValue)
				*d_pDerivativeOfLastOutputThisTest2 = dDerivativeOfLastOutput2;

			PRINT_MEMORY_INFO(dp_pDerivativeOfLastOutput,d_pDerivativeOfLastOutputThisTest);
		}
	}
}

extern "C" void executeLayerCUDA(const real_gpu *dp_pLayerInput,const real_gpu *dp_pWeights,real_gpu *dp_pLayerOutput,real_gpu *dp_pDerivativeOfLastOutput
								 ,int p_iTestCount,int p_iOutputNeuronCount,int p_iNumInputNeurons,Neuron::NeuronType p_eNeuronType,const int *p_pVecTestIndices)
{
	// blockDim.x should be a multiple of 16 (half warp). We will be able to retrieve global data using coalescing
	int iBlockDimUpdated = ALIGN_UP(p_iOutputNeuronCount+1,HALF_WARP);
	int iSharedMemorySize = 2 * p_iNumInputNeurons * sizeof(real_gpu); // memory for input

	int iNumOfWeights = p_iNumInputNeurons * p_iOutputNeuronCount;
	int iNumOfWeightsAligned = ALIGN_UP(iNumOfWeights,iBlockDimUpdated);
	int iMaxNumberOfSimulatenousBlocks = 512 / iBlockDimUpdated + ((512 % iBlockDimUpdated) != 0);

	int iMaxMemPerBlock = 0;
	iMaxMemPerBlock = max(0,(iMaxNumberOfSharedMemoryElementsForWeights / iMaxNumberOfSimulatenousBlocks - 2 * p_iNumInputNeurons));

	int iHowMuchMemoryForWeights = (min(iNumOfWeightsAligned,max(512,iMaxMemPerBlock)) / iBlockDimUpdated) * iBlockDimUpdated;

	iSharedMemorySize += iHowMuchMemoryForWeights * sizeof(real_gpu); // memory for weights

	// If p_pVecTestIndices!=NULL , then we use constant memory to set test indices for the kernel
	if(p_pVecTestIndices!=NULL)
	{
		hipMemcpyToSymbol(HIP_SYMBOL("iTestIndices"),p_pVecTestIndices,p_iTestCount*sizeof(int),0);
	}

	int iNumInputNeuronsAligned = ALIGN_UP(p_iNumInputNeurons, HALF_WARP);

	executeLayerKernel <<<(p_iTestCount+1)/2,iBlockDimUpdated,iSharedMemorySize>>> (dp_pLayerInput,dp_pWeights,dp_pLayerOutput,dp_pDerivativeOfLastOutput,p_iNumInputNeurons
		,iNumInputNeuronsAligned,p_eNeuronType,p_iOutputNeuronCount,(p_pVecTestIndices!=NULL),iHowMuchMemoryForWeights,p_iTestCount);
}


__global__ void calculateErrorInLastLayerKernel(const real_gpu *dp_pCorrectOutput,const real_gpu *dp_pNetworkOutput,real_gpu *dp_pErrors,int p_iSpaceBetweenTestsInOutput)
{
	int iElementIndexCorrectOutput = p_iSpaceBetweenTestsInOutput * iTestIndices[blockIdx.x] + threadIdx.x;
	int iElementIndex = p_iSpaceBetweenTestsInOutput * blockIdx.x + threadIdx.x;
	dp_pErrors[iElementIndex] = dp_pNetworkOutput[iElementIndex] - dp_pCorrectOutput[iElementIndexCorrectOutput];
	PRINT_DEBUG_INFO("GPU: Test in batch nr %d (test %d) , Output %d (iElementIndex %d) : Network = %f , Correct  = %f , Error = %f\n",blockIdx.x,iTestIndices[blockIdx.x],threadIdx.x
		,iElementIndex,dp_pNetworkOutput[iElementIndex],dp_pCorrectOutput[iElementIndexCorrectOutput],dp_pErrors[iElementIndex]);
	PRINT_MEMORY_INFO(dp_pErrors,&dp_pErrors[iElementIndex]);
	PRINT_MEMORY_INFO(dp_pNetworkOutput,&dp_pNetworkOutput[iElementIndex]);
	PRINT_MEMORY_INFO(dp_pCorrectOutput,&dp_pCorrectOutput[iElementIndexCorrectOutput]);
}

extern "C" void calculateErrorInLastLayerCUDA(const real_gpu *dp_pCorrectOutput,const real_gpu *dp_pNetworkOutput,real_gpu *dp_pErrors,int p_iOutputNeuronCount,int p_iNumTestsInBatch,int p_iSpaceBetweenTestsInOutput)
{
	calculateErrorInLastLayerKernel <<<p_iNumTestsInBatch,p_iOutputNeuronCount>>> (dp_pCorrectOutput,dp_pNetworkOutput,dp_pErrors,p_iSpaceBetweenTestsInOutput);
}


__global__ void calculateErrorInNotLastLayerKernel(const real_gpu *dp_pNextLayerWeights,const real_gpu *dp_pNextLayerError,real_gpu *dp_pThisLayerError,int p_iNextLayerNeuronCount
												   ,int p_iNextLayerNeuronCountAligned,int p_iThisLayerNeuronCount,int p_iNumTestsInBatch)
{
	extern __shared__ real_gpu s_NextLayerErrorThisTest[];
	real_gpu* s_NextLayerErrorThisTest2 = &s_NextLayerErrorThisTest[p_iNextLayerNeuronCount];
	real_gpu* s_NextLayerWeights = &s_NextLayerErrorThisTest2[p_iNextLayerNeuronCount];
	real_gpu dError = 0.0f;
	real_gpu dError2 = 0.0f;
	int iNextLayerWeightsForOneNeuron = p_iThisLayerNeuronCount + 1;
	const real_gpu *d_pNextLayerErrorThisTest = dp_pNextLayerError + p_iNextLayerNeuronCountAligned * (2*blockIdx.x);
	const real_gpu *d_pNextLayerErrorThisTest2 = dp_pNextLayerError + p_iNextLayerNeuronCountAligned * (2*blockIdx.x+1);

	// Copying error data from global to shared memory
	for(int iErrorIndex = threadIdx.x;iErrorIndex < p_iNextLayerNeuronCount; iErrorIndex += blockDim.x)
	{
		s_NextLayerErrorThisTest[iErrorIndex] = d_pNextLayerErrorThisTest[iErrorIndex];
		s_NextLayerErrorThisTest2[iErrorIndex] = d_pNextLayerErrorThisTest2[iErrorIndex];
		PRINT_MEMORY_INFO(dp_pNextLayerError,&d_pNextLayerErrorThisTest[iErrorIndex]);
	}

	__syncthreads();

	// I can't check thread index, because later I use __syncthreads() ...
	//if(threadIdx.x < p_iThisLayerNeuronCount)
	{
		for(int iWeightIndex = 0;iWeightIndex < p_iNextLayerNeuronCount; ++iWeightIndex)
		{
			PRINT_DEBUG_INFO("GPU: Test index %d , Neuron index %d , Weight index %d : dp_pNextLayerWeights [%d] = %f , dp_pNextLayerError[%d] = %f , MULT = %f\n"
				,blockIdx.x,threadIdx.x,iWeightIndex,iWeightIndex*iNextLayerWeightsForOneNeuron + threadIdx.x,dp_pNextLayerWeights[iWeightIndex*iNextLayerWeightsForOneNeuron + threadIdx.x],iWeightIndex
				,dp_pNextLayerError[iWeightIndex],dp_pNextLayerWeights[iWeightIndex*iNextLayerWeightsForOneNeuron + threadIdx.x] * dp_pNextLayerError[iWeightIndex]);

			// we load weights twice - in case the first loaded weight position is not divisible by HALF_WARP
			int iWeightFirstAddress = iWeightIndex*iNextLayerWeightsForOneNeuron;
			int iFirstAddressToLoad = (iWeightFirstAddress / HALF_WARP) * HALF_WARP;
			s_NextLayerWeights[threadIdx.x] = dp_pNextLayerWeights[iFirstAddressToLoad + threadIdx.x];
			s_NextLayerWeights[blockDim.x + threadIdx.x] = dp_pNextLayerWeights[iFirstAddressToLoad + blockDim.x + threadIdx.x];
			__syncthreads();

			int iWeightIndexInSharedMemory = iWeightFirstAddress - iFirstAddressToLoad + threadIdx.x;
			dError += s_NextLayerWeights[iWeightIndexInSharedMemory] * s_NextLayerErrorThisTest[iWeightIndex];
			dError2 += s_NextLayerWeights[iWeightIndexInSharedMemory] * s_NextLayerErrorThisTest2[iWeightIndex];
			PRINT_MEMORY_INFO(dp_pNextLayerWeights,&dp_pNextLayerWeights[iWeightIndex*iNextLayerWeightsForOneNeuron + threadIdx.x]);

			__syncthreads();
		}

		if(threadIdx.x < p_iThisLayerNeuronCount)
		{
			dp_pThisLayerError[blockDim.x*(2*blockIdx.x) + threadIdx.x] = dError;

			if(2*blockIdx.x != p_iNumTestsInBatch-1)
				dp_pThisLayerError[blockDim.x*(2*blockIdx.x+1) + threadIdx.x] = dError2;

			PRINT_MEMORY_INFO(dp_pThisLayerError,&dp_pThisLayerError[blockDim.x*blockIdx.x + threadIdx.x]);
			PRINT_DEBUG_INFO("GPU: Test index %d , Neuron index %d : Error %f\n",blockIdx.x,threadIdx.x,dError);
		}
	}
}

extern "C" void calculateErrorInNotLastLayerCUDA(const real_gpu *dp_pNextLayerWeights,const real_gpu *dp_pNextLayerError,real_gpu *dp_pThisLayerError,int p_iThisLayerNeuronCount,int p_iNextLayerNeuronCount,int p_iNumTestsInBatch)
{
	int iElementsAllocatedForOneTestInNextLayerAligned = ALIGN_UP(p_iNextLayerNeuronCount+1,HALF_WARP);
	int iElementsAllocatedForOneTestInThisLayerAligned = ALIGN_UP(p_iThisLayerNeuronCount+1,HALF_WARP);
	int iSharedMemorySize = 2 * p_iNextLayerNeuronCount * sizeof(real_gpu); // memory for error
	iSharedMemorySize += 2 * iElementsAllocatedForOneTestInThisLayerAligned * sizeof(real_gpu); // memory for weights

	calculateErrorInNotLastLayerKernel <<<(p_iNumTestsInBatch+1)/2,iElementsAllocatedForOneTestInThisLayerAligned,iSharedMemorySize>>>
		(dp_pNextLayerWeights,dp_pNextLayerError,dp_pThisLayerError,p_iNextLayerNeuronCount,iElementsAllocatedForOneTestInNextLayerAligned,p_iThisLayerNeuronCount,p_iNumTestsInBatch);
}


__global__ void updateWeightsInTrainingKernel(const real_gpu *dp_pThisLayerError,const real_gpu *dp_pDerivativeOfLastOutput,const real_gpu *dp_pLayerBeforeOutputs
											  ,real_gpu p_dEta,real_gpu *dp_pThisLayerWeights,int p_iNumTestsInBatch,int iElementsAllocatedForOneTestInThisLayerAligned
											  ,int p_iElementsAllocatedForOneTestInLayerBeforeAligned,bool p_bLayerBeforeOutputsHaveSpecificIndexes,int p_iThisLayerNeuronCount)
{
	// We change: neuron blockIdx.x , weight threadIdx.x
	extern __shared__ real_gpu s_ThisLayerError[];
	real_gpu* s_DerivativeOfLastOutput = &s_ThisLayerError[2*p_iNumTestsInBatch];

	// Two first threads in each block copy global memory to shared memory
	if(threadIdx.x == 0 || threadIdx.x == 1)
	{
		for(unsigned uTestIndex = 0;uTestIndex < p_iNumTestsInBatch;++uTestIndex)
		{
			s_ThisLayerError[2*uTestIndex+threadIdx.x] = dp_pThisLayerError[iElementsAllocatedForOneTestInThisLayerAligned*uTestIndex + 2*blockIdx.x + threadIdx.x];
			s_DerivativeOfLastOutput[2*uTestIndex+threadIdx.x] = dp_pDerivativeOfLastOutput[iElementsAllocatedForOneTestInThisLayerAligned*uTestIndex + 2*blockIdx.x + threadIdx.x];
		}
	}

	__syncthreads();

	real_gpu dChange = 0.0f;
	real_gpu dChange2 = 0.0f;
	for(unsigned uTestIndex = 0;uTestIndex < p_iNumTestsInBatch;++uTestIndex)
	{
		int iTestIndexForOutputBefore = ( p_bLayerBeforeOutputsHaveSpecificIndexes ? iTestIndices[uTestIndex] : uTestIndex );
		real_gpu dLayerBeforeOutput = dp_pLayerBeforeOutputs[p_iElementsAllocatedForOneTestInLayerBeforeAligned*iTestIndexForOutputBefore + threadIdx.x];

		real_gpu dChangeThisTest = s_ThisLayerError[2*uTestIndex] * s_DerivativeOfLastOutput[2*uTestIndex] * dLayerBeforeOutput * p_dEta;
		real_gpu dChangeThisTest2 = s_ThisLayerError[2*uTestIndex+1] * s_DerivativeOfLastOutput[2*uTestIndex+1] * dLayerBeforeOutput * p_dEta;
		PRINT_DEBUG_INFO("GPU: Test %d , Neuron %d , Weight %d : dLayerBeforeOutput %f , dChangeThisTest %f\n",uTestIndex,blockIdx.x,threadIdx.x,dLayerBeforeOutput,dChangeThisTest);
		dChange += dChangeThisTest;
		dChange2 += dChangeThisTest2;

		PRINT_MEMORY_INFO(dp_pThisLayerError,&dp_pThisLayerError[iElementsAllocatedForOneTestInThisLayerAligned*uTestIndex + blockIdx.x]);
		PRINT_MEMORY_INFO(dp_pDerivativeOfLastOutput,&dp_pDerivativeOfLastOutput[iElementsAllocatedForOneTestInThisLayerAligned*uTestIndex + blockIdx.x]);
		PRINT_MEMORY_INFO(dp_pLayerBeforeOutputs,&dp_pLayerBeforeOutputs[p_iElementsAllocatedForOneTestInLayerBeforeAligned*iTestIndexForOutputBefore + threadIdx.x]);
	}

	//int iTestIndexForWeights = ( p_bLayerBeforeOutputsHaveSpecificIndexes ? iTestIndices[blockIdx.x] : blockIdx.x );
	int iWeightIndex = blockDim.x*(2*blockIdx.x) + threadIdx.x;
	int iWeightIndex2 = blockDim.x*(2*blockIdx.x+1) + threadIdx.x;
	PRINT_DEBUG_INFO("GPU: Neuron %d , Weight %d (index in array %d) : Old weight %f , Change %f , New weight %f\n",blockIdx.x,threadIdx.x,iWeightIndex,dp_pThisLayerWeights[iWeightIndex],dChange,dp_pThisLayerWeights[iWeightIndex] - dChange);
	dp_pThisLayerWeights[iWeightIndex] = dp_pThisLayerWeights[iWeightIndex] - dChange;

	if(2*blockIdx.x != p_iThisLayerNeuronCount-1)
		dp_pThisLayerWeights[iWeightIndex2] = dp_pThisLayerWeights[iWeightIndex2] - dChange2;

	PRINT_MEMORY_INFO(dp_pThisLayerWeights,&dp_pThisLayerWeights[iWeightIndex]);
}

extern "C" void updateWeightsInTrainingCUDA(const real_gpu *dp_pThisLayerError,const real_gpu *dp_pDerivativeOfLastOutput,const real_gpu *dp_pLayerBeforeOutputs,real_gpu p_dEta,int p_iThisLayerNeuronCount
											,int p_iNumOutputsLayerBefore,real_gpu *dp_pThisLayerWeights,int p_iNumTestsInBatch,bool p_bLayerBeforeOutputsHaveSpecificIndexes)
{
	int iElementsAllocatedForOneTestInThisLayerAligned = ALIGN_UP(p_iThisLayerNeuronCount+1,HALF_WARP);
	int iElementsAllocatedForOneTestInLayerBeforeAligned = ALIGN_UP(p_iNumOutputsLayerBefore+1,HALF_WARP);
	int iSharedMemorySize = 4 * p_iNumTestsInBatch * sizeof(real_gpu); // memory for ThisLayerError and DerivativeOfLastOutput (2 times)

	updateWeightsInTrainingKernel <<<(p_iThisLayerNeuronCount+1)/2,p_iNumOutputsLayerBefore+1,iSharedMemorySize>>> (dp_pThisLayerError,dp_pDerivativeOfLastOutput,dp_pLayerBeforeOutputs,p_dEta
		,dp_pThisLayerWeights,p_iNumTestsInBatch,iElementsAllocatedForOneTestInThisLayerAligned,iElementsAllocatedForOneTestInLayerBeforeAligned,p_bLayerBeforeOutputsHaveSpecificIndexes,p_iThisLayerNeuronCount);
}
